#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>

#include "Space.h"

#include "Vector.h"
#include "Molecule.h"
#include "hip/hip_runtime.h"

#define Boltzmann  1.3806488e-23d
#define Angstrom  10e-10d
#define AtomicMassUnit  1.660538921e-27d
#define sigma 2.74*10e-10	//m
#define epsilon 36.2*1.3806488e-23	//J

#define maxDistSquare 4.0 * sigma * 4.0 * sigma;




typedef std::vector<Molecule> MoloculesList;

const size_t maxSize = 1024 * 16;
double h_newF[3 * maxSize], h_r[3 * maxSize];
double *d_newF, *d_r, *d_squares;
double *space;


void allocateMemory(size_t size)
{
	if (size > maxSize)
		throw 0;		//WARNING
	hipMalloc(&d_newF, sizeof(double) * 3 * size);
	hipMalloc(&d_r, sizeof(double) * 3 * size);
	hipMalloc(&d_squares, sizeof(double) * size);
}
void freeMemory()
{
	hipFree(d_newF);
	hipFree(d_r);
	hipFree(d_squares);
}

__global__ void kernel_calculateSquares(double *squares, const double *r, int n)
{
	auto lambdaTest = [&]() -> int {
		return blockDim.x * blockIdx.x + threadIdx.x;
	};
	int molecule = lambdaTest();//blockDim.x * blockIdx.x + threadIdx.x;
	const double sigmaSquare = sigma * sigma;
	if (molecule >= n) return;
	int rIndex = molecule * 3;
	double sqX = r[rIndex + 0] * r[rIndex + 0];
	double sqY = r[rIndex + 1] * r[rIndex + 1];
	double sqZ = r[rIndex + 2] * r[rIndex + 2];
	double sqR = sqX + sqY + sqZ;		//sqR = r^2
	if (sqR == 0) {
		squares[molecule] = 0;
		return;
	}
	double sq = sigmaSquare / sqR;		//sq = (sigma/r)^2
	squares[molecule] = sq*sq*sq*sq;			// = (sigma/r)^8
}


__global__ void kernel_calculateNewForces_GPU(double *forces, const double *r, const double *squares, int n)
{
	int molecule = blockDim.x * blockIdx.x + threadIdx.x;
	int component = blockDim.y * blockIdx.y + threadIdx.y;
	if (molecule >= n) return;
	//if (component >= 3) return;
	const double c = -(24.0 * epsilon) / (sigma * sigma);
	int componetIndex = molecule * 3 + component;
	double sq = squares[molecule];
	forces[componetIndex] += c * ((2 * sq*sq) - (sq)) * r[componetIndex];
}





void calculateNewForces_GPU(MoloculesList &molecules1, MoloculesList &molecules2)
{
	auto end1 = molecules1.end();
	for (auto i = molecules1.begin(); i != end1; ++i) {
		auto end2 = molecules2.end();

		//copy data to C-array
		size_t index = 0;
		for (auto j = molecules2.begin(); j != end2; ++j) {
			h_r[index++] = (*j).r.x;
			h_r[index++] = (*j).r.y;
			h_r[index++] = (*j).r.z;
		}

		//allocate memory on GPU
		size_t size = molecules2.size();
		//if (size > maxSize)
		//	throw 0;		//WARNING
		//double *d_newF, *d_r, *d_squares;
		//hipMalloc(&d_newF, sizeof(double) * 3 * size);
		//hipMalloc(&d_r, sizeof(double) * 3 * size);
		//hipMalloc(&d_squares, sizeof(double) * size);

		//copy array to GPU
		hipMemcpy(d_r, h_r, 3 * sizeof(double)*size, hipMemcpyHostToDevice);

		//calculations
		int numberOfCores = 512;
		dim3 grid, blocks;
		grid = dim3(size / numberOfCores + 1, 1, 1);
		blocks = dim3(numberOfCores, 1, 1);
		kernel_calculateSquares << <grid, blocks >> >(d_squares, d_r, size);
		hipDeviceSynchronize();
		blocks = dim3(numberOfCores, 3, 1);
		kernel_calculateNewForces_GPU << <grid, blocks >> >(d_newF, d_r, d_squares, size);
		hipDeviceSynchronize();

		//copy array from GPU;
		hipMemcpy(h_r, d_r, 3 * sizeof(double)*size, hipMemcpyDeviceToHost);

		//copy C-array to data
		index = 0;
		for (auto j = molecules2.begin(); j != end2; ++j) {
			(*j).r.x = h_r[index++];
			(*j).r.y = h_r[index++];
			(*j).r.z = h_r[index++];
		}

		//free memory on GPU
		//hipFree(d_newF);
		//hipFree(d_r);
		//hipFree(d_squares);

	}
}


/*
__device__ inline Vector Force_LennardJones_GPU(Vector r, double square)
{
	register const double sigmaSquare = sigma * sigma;
	square = sigmaSquare / square;
	register double U = 2.0*pow(square, 14 / 2) - pow(square, 8 / 2);
	register const double c = -24.0 * epsilon / sigmaSquare;
	return c * U * r;
}

__device__ void calculateNewForces_GPU_rutine(MoloculesList &molecules1, MoloculesList &molecules2)
{
	auto end1 = molecules1.end();
	for (auto i = molecules1.begin(); i != end1; ++i) {
		auto end2 = molecules2.end();
		for (auto j = molecules2.begin(); j != end2; ++j) {
			if (i._Ptr == j._Ptr) continue;
			Vector r = (*j).r - (*i).r;
			register double square = r.square();
			//if (maxDistSquare < square) continue;
			(*i).newF += Force_LennardJones_GPU(r, square);
		}
	}
}

__global__ void calculateNewForcesForUnderspace_GPU(Space* space, int nx, int ny, int nz)
{
	//QThread::msleep(1);
	Underspace &centralSpace = space->underspaces[nx][ny][nz];
	auto closestSpaces = { -1, 0, 1 };
	for (const auto &dx : closestSpaces) {
		for (const auto &dy : closestSpaces) {
			for (const auto &dz : closestSpaces) {
				int x = nx + dx;
				int y = ny + dy;
				int z = nz + dz;
				if (x < 0 || y < 0 || z < 0) continue;
				if (x >= space->Nx) continue;
				if (y >= space->Ny) continue;
				if (z >= space->Nz) continue;
				calculateNewForces_GPU_rutine(centralSpace.molecules, space->underspaces[x][y][z].molecules);
			}
		}
	}
}*/



