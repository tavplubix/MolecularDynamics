#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>



#include "Vector.h"
#include "Molecule.h"
#include "hip/hip_runtime.h"

#define Boltzmann  1.3806488e-23d
#define Angstrom  10e-10d
#define AtomicMassUnit  1.660538921e-27d
#define sigma 2.74*10e-10	//m
#define epsilon 36.2*1.3806488e-23	//J





typedef std::vector<Molecule> MoloculesList;

const size_t maxSize = 1024 * 16;
double h_newF[3 * maxSize], h_r[3 * maxSize];



__global__ void kernel_calculateSquares(double *squares, const double *r, int n)
{
	int molecule = blockDim.x * blockIdx.x + threadIdx.x;
	const double sigmaSquare = sigma * sigma;
	if (molecule >= n) return;
	int rIndex = molecule * 3;
	double sqX = r[rIndex + 0] * r[rIndex + 0];
	double sqY = r[rIndex + 1] * r[rIndex + 1];
	double sqZ = r[rIndex + 2] * r[rIndex + 2];
	double sqR = sqX + sqY + sqZ;		//sqR = r^2
	if (sqR == 0) {
		squares[molecule] = 0;
		return;
	}
	double sq = sigmaSquare / sqR;		//sq = (sigma/r)^2
	squares[molecule] = sq*sq*sq*sq;			// = (sigma/r)^8
}


__global__ void kernel_calculateNewForces_GPU(double *forces, const double *r, const double *squares, int n)
{
	int molecule = blockDim.x * blockIdx.x + threadIdx.x;
	int component = blockDim.y * blockIdx.y + threadIdx.y;
	if (molecule >= n) return;
	//if (component >= 3) return;
	const double c = -(24.0 * epsilon) / (sigma * sigma);
	int componetIndex = molecule * 3 + component;
	double sq = squares[molecule];
	forces[componetIndex] += c * ((2 * sq*sq) - (sq)) * r[componetIndex];
}





void calculateNewForces_GPU(MoloculesList &molecules1, MoloculesList &molecules2)
{
	auto end1 = molecules1.end();
	for (auto i = molecules1.begin(); i != end1; ++i) {
		auto end2 = molecules2.end();

		//copy data to C-array
		size_t index = 0;
		for (auto j = molecules2.begin(); j != end2; ++j) {
			h_r[index++] = (*j).r.x;
			h_r[index++] = (*j).r.y;
			h_r[index++] = (*j).r.z;
		}

		//allocate memory on GPU
		size_t size = molecules2.size();
		if (size > maxSize)
			throw 0;		//WARNING
		double *d_newF, *d_r, *d_squares;
		hipMalloc(&d_newF, sizeof(double) * 3 * size);
		hipMalloc(&d_r, sizeof(double) * 3 * size);
		hipMalloc(&d_squares, sizeof(double) * size);

		//copy array to GPU
		hipMemcpy(d_r, h_r, 3 * sizeof(double)*size, hipMemcpyHostToDevice);

		//calculations
		int numberOfCores = 512;
		dim3 grid, blocks;
		grid = dim3(size / numberOfCores + 1, 1, 1);
		blocks = dim3(numberOfCores, 1, 1);
		kernel_calculateSquares << <grid, blocks >> >(d_squares, d_r, size);
		hipDeviceSynchronize();
		blocks = dim3(numberOfCores, 3, 1);
		kernel_calculateNewForces_GPU << <grid, blocks >> >(d_newF, d_r, d_squares, size);
		hipDeviceSynchronize();

		//copy array from GPU;
		hipMemcpy(h_r, d_r, 3 * sizeof(double)*size, hipMemcpyDeviceToHost);

		//copy C-array to data
		index = 0;
		for (auto j = molecules2.begin(); j != end2; ++j) {
			(*j).r.x = h_r[index++];
			(*j).r.y = h_r[index++];
			(*j).r.z = h_r[index++];
		}

		//free memory on GPU
		hipFree(d_newF);
		hipFree(d_r);
		hipFree(d_squares);

	}
}










