#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <iostream>

#include "c_cuda_structures.h"



//==========================================================================
//					Operations with CUDAVector
//==========================================================================

__device__ void mov(const CUDAVector &a, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i];
}

__device__ void add(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] + b.v[i];
}

__device__ void sub(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] - b.v[i];
}

__device__ void mul(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] * b.v[i];
}

__device__ void mul(const CUDAVector &a, const myfloat b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] * b;
}

__device__ void div(const CUDAVector &a, const myfloat b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] / b;
}

__device__ double square(const CUDAVector &a)
{
	double result = 0;
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result += a.v[i] * a.v[i];
	return result;
}


//==========================================================================
//					Operations with CUDAMolecule
//==========================================================================

#define Boltzmann  1.3806488e-23
#define Angstrom  1e-10
#define AtomicMassUnit  1.660538921e-27
#define sigma (2.74*1e-10)	//m
#define epsilon (36.2*1.3806488e-23)	//J
#define mass (20.1797 * 1.660538921e-27)

#define maxDistSquare (4.0 * sigma * 4.0 * sigma)


__device__ inline void d_Force_LennardJones(const CUDAVector& r, myfloat square, CUDAVector& F)
{
	const myfloat sigmaSquare = sigma * sigma;
	square = sigmaSquare / square;
	myfloat U = 2.0*pow(square, 14 / 2) - pow(square, 8 / 2);
	const myfloat c = -24.0 * epsilon / sigmaSquare;
	mul(r, c * U, F);
}


//==========================================================================
//					Operations with CUDAUnderspace
//==========================================================================


__device__ void d_recalculatePositions_Beeman(CUDAUnderspace *cus, myfloat dt)
{
	auto molecules = GET_POINTER(CUDAMolecule, cus, cus->moleculesShift);
	for (size_t i = 0; i < cus->numberOfMolecules; ++i) {
		mov(molecules[i].r, molecules[i].oldr);
		CUDAVector tmp;
		//i.r += i.v * dt;
		mul(molecules[i].v, dt, tmp);
		add(molecules[i].r, tmp, molecules[i].r);
		//i.r += 4.0 / 6.0 * (i.F / i.m) * (dt*dt);
		mul(molecules[i].F, (4.0 / 6.0) * (dt*dt) / mass, tmp);
		add(molecules[i].r, tmp, molecules[i].r);
		//i.r -= 1.0 / 6.0 * (i.oldF / i.m) * (dt*dt);
		mul(molecules[i].oldF, - (1.0/6.0) * (dt*dt) / mass, tmp);
		add(molecules[i].r, tmp, molecules[i].r);
	}
}

__device__ void d_recalculateSpeeds_Beeman(CUDAUnderspace *cus, myfloat dt, int width, int height)
{
	auto molecules = GET_POINTER(CUDAMolecule, cus, cus->moleculesShift);
	for (size_t i = 0; i < cus->numberOfMolecules; ++i) {
		//cus->molecules[i].newF.v[0] += 1e10;
		CUDAVector tmp;
		//i.v += 2.0 / 6.0 * (i.newF / i.m) * dt;
		mul(molecules[i].newF, (2.0 / 6.0) * dt / mass, tmp);
		add(molecules[i].v, tmp, molecules[i].v);
		//i.v += 5.0 / 6.0 * (i.F / i.m) * dt;
		mul(molecules[i].F, (5.0 / 6.0) * dt / mass, tmp);
		add(molecules[i].v, tmp, molecules[i].v);
		//i.v -= 1.0 / 6.0 * (i.oldF / i.m) * dt;
		mul(molecules[i].oldF, - (1.0 / 6.0) * dt / mass, tmp);
		add(molecules[i].v, tmp, molecules[i].v);

		if (molecules[i].r.v[0] <= 0) {
			molecules[i].v.v[0] = abs(molecules[i].v.v[0]);
		}
		if (width * Angstrom <= molecules[i].r.v[0]) {
			molecules[i].v.v[0] = -abs(molecules[i].v.v[0]);
		}

		if (molecules[i].r.v[1] <= 0) {
			molecules[i].v.v[1] = abs(molecules[i].v.v[1]);
		}
		if (height * Angstrom <= molecules[i].r.v[1]) {
			molecules[i].v.v[1] = -abs(molecules[i].v.v[1]);
		}
	}
}

__device__ void d_calculateNewForcesForUnderspace(CUDASpace *cs, int nx, int ny, int nz)
{
	auto underspaces = GET_POINTER(CUDAUnderspace, cs, cs->underspacesShift);
	CUDAUnderspace *centralSpace = &underspaces[LINEAR(cs, nx, ny, nz)];
	int  closest = 1;
	for (int dx = -closest; dx <= closest; ++dx) {
		for (int dy = -closest; dy <= closest; ++dy) {
			for (int dz = -closest; dz <= closest; ++dz) {
				int x = nx + dx;
				int y = ny + dy;
				int z = nz + dz;
				if (x < 0 || y < 0 || z < 0) continue;
				if (x >= cs->Nx) continue;
				if (y >= cs->Ny) continue;
				if (z >= cs->Nz) continue;
				d_calculateNewForces(centralSpace, &underspaces[LINEAR(cs, x, y, z)]);
			}
		}
	}
}

__device__ void d_calculateNewForces(CUDAUnderspace *cus1, CUDAUnderspace *cus2)
{
	auto molecules1 = GET_POINTER(CUDAMolecule, cus1, cus1->moleculesShift);
	auto molecules2 = GET_POINTER(CUDAMolecule, cus2, cus2->moleculesShift);
	for (size_t i = 0; i < cus1->numberOfMolecules; ++i) {
		for (size_t j = 0; j < cus2->numberOfMolecules; ++j) {
			//if (i == j && cus1 == cus2) continue;
			//Vector r = (*j).r - (*i).r;
			CUDAVector tmp;
			sub(molecules2[j].r, molecules1[i].r, tmp);
			myfloat sq = square(tmp);
			if (sq == 0) continue;
			if (maxDistSquare < sq) continue;
			//(*i).newF += d_Force_LennardJones(r, sq);
			d_Force_LennardJones(tmp, sq, tmp);
			add(molecules1[i].newF, tmp, molecules1[i].newF);
		}
	}
}


//==========================================================================
//					Operations with CUDASpace
//==========================================================================

__global__ void cuda_recalculatePositions(CUDASpace *cs)
{
	auto underspaces = GET_POINTER(CUDAUnderspace, cs, cs->underspacesShift);
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (nz >= cs->Nz) return;

	d_recalculatePositions_Beeman(&underspaces[LINEAR(cs, nx, ny, nz)], cs->dt);
}

__global__ void cuda_recalculateSpeeds(CUDASpace *cs)
{
	auto underspaces = GET_POINTER(CUDAUnderspace, cs, cs->underspacesShift);
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (nz >= cs->Nz) return;

	d_recalculateSpeeds_Beeman(&underspaces[LINEAR(cs, nx, ny, nz)], cs->dt, cs->width, cs->height);
}

__global__ void cuda_recalculateForces(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;	
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (ny >= cs->Ny) return;

	d_calculateNewForcesForUnderspace(cs, nx, ny, nz);
}

__global__ void cuda_validate(CUDASpace *cs)
{
	auto underspaces = GET_POINTER(CUDAUnderspace, cs, cs->underspacesShift);
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (nz >= cs->Nz) return;

	CUDAUnderspace& cus = underspaces[LINEAR(cs, nx, ny, nz)];
	auto molecules = GET_POINTER(CUDAMolecule, &cus, cus.moleculesShift);
	for (size_t i = 0; i < cus.numberOfMolecules; ++i) {
		//t.oldF = t.F;
		mov(molecules[i].F, molecules[i].oldF);
		//t.F = t.newF;
		mov(molecules[i].newF, molecules[i].F);
	}
}

__global__ void cuda_dropNewF(CUDASpace *cs)
{
	auto underspaces = GET_POINTER(CUDAUnderspace, cs, cs->underspacesShift);
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (nz >= cs->Nz) return;
	CUDAUnderspace& cus = underspaces[LINEAR(cs, nx, ny, nz)];
	auto molecules = GET_POINTER(CUDAMolecule, &cus, cus.moleculesShift);
	for (size_t i = 0; i < cus.numberOfMolecules; ++i) {
		//t.newF = Vector();
		molecules[i].newF.v[0] = 0;
		molecules[i].newF.v[1] = 0;
		molecules[i].newF.v[2] = 0;
	}
}

void cuda_oneStep(CUDASpace *d_cs,int Nx, int Ny, int Nz)
{
	int numberOfCores = 1024;
	//int coresPerDim = int(pow(numberOfCores, 1.0/3.0));
	int coresPerDim = int(sqrt(numberOfCores));
	dim3 grid, blocks;
	//grid = dim3(Nx / coresPerDim + 1, Ny / coresPerDim + 1, Nz / coresPerDim + 1);
	grid = dim3(Nx / coresPerDim + 1, Ny / coresPerDim + 1, Nz);
	blocks = dim3(coresPerDim, coresPerDim, 1);

	cuda_recalculatePositions	<<<grid, blocks>>> (d_cs);
	//auto cudaStatus = hipGetLastError();
	hipDeviceSynchronize();

	cuda_dropNewF				<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();

	cuda_recalculateForces		<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();

	cuda_recalculateSpeeds		<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();

	cuda_validate				<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();
}



extern CUDASpace* moveFromHost(CUDASpace *h_cs, size_t wholeSize/* = 0*/)
{
	CUDASpace *d_cs;
	if (wholeSize == 0) 
		wholeSize = WHOLE_SIZE_OF_SPACE(h_cs);
	hipMalloc(&d_cs, wholeSize);		//allocate device memory for all data
	hipMemcpy(d_cs, h_cs, wholeSize, hipMemcpyHostToDevice);		//copy all data from h_cs (host) to d_cs (device)
	//delete[] reinterpret_cast<byte*>(h_cs);		//delete data from host memory;

	return d_cs;
}

CUDASpace* moveFromDevice(CUDASpace *d_cs, size_t wholeSize/* = 0*/, byte *h_p/* = nullptr*/)
{
	if (wholeSize == 0) 
		throw 0;		//TODO copy CUDASpace only from device and calculate wholeSize

	if (h_p == nullptr) 
		h_p = new byte[wholeSize];		//allocate host memory for all data
	hipMemcpy(h_p, d_cs, wholeSize, hipMemcpyDeviceToHost);	//copy all data
	auto h_cs = reinterpret_cast<CUDASpace*>(h_p);		//get pointer to CUDASpace
	hipFree(d_cs);		//delete data from device memory

	return h_cs;
}

/*
void freeDeviceMem(CUDASpace *d_cs)
{
	for (size_t i = 0; i < d_cs->Nx; ++i) {
		for (size_t j = 0; j < d_cs->Ny; ++j) {
			for (size_t k = 0; k < d_cs->Nz; ++k) {
				hipFree(d_cs->underspaces[i][j][k].molecules);
			}
			hipFree(d_cs->underspaces[i][j]);
		}
		hipFree(d_cs->underspaces[i]);
	}
	hipFree(d_cs->underspaces);
	hipFree(d_cs);
}

void freeHostMem(CUDASpace *h_cs)
{
	for (size_t i = 0; i < h_cs->Nx; ++i) {
		for (size_t j = 0; j < h_cs->Ny; ++j) {
			for (size_t k = 0; k < h_cs->Nz; ++k) {
				delete h_cs->underspaces[i][j][k].molecules;
			}
			delete h_cs->underspaces[i][j];
		}
		delete h_cs->underspaces[i];
	}
	delete h_cs->underspaces;
	delete h_cs;
}
*/

