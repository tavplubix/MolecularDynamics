#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <iostream>

#include "c_cuda_structures.h"



//==========================================================================
//					Operations with CUDAVector
//==========================================================================

__device__ void mov(const CUDAVector &a, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i];
}

__device__ void add(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] + b.v[i];
}

__device__ void sub(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] - b.v[i];
}

__device__ void mulv(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] * b.v[i];
}

__device__ void mulc(const CUDAVector &a, const double b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] * b;
}

__device__ void div(const CUDAVector &a, const double b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] / b;
}

__device__ double square(const CUDAVector &a)
{
	double result = 0;
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result += a.v[i] * a.v[i];
	return result;
}


//==========================================================================
//					Operations with CUDAMolecule
//==========================================================================

#define Boltzmann  1.3806488e-23
#define Angstrom  1e-10
#define AtomicMassUnit  1.660538921e-27
#define sigma (2.74*1e-10)	//m
#define epsilon (36.2*1.3806488e-23)	//J
#define mass (20.1797 * 1.660538921e-27)

#define maxDistSquare (4.0 * sigma * 4.0 * sigma)


__device__ inline void d_Force_LennardJones(const CUDAVector& r, double square, CUDAVector& F)
{
	const double sigmaSquare = sigma * sigma;
	square = sigmaSquare / square;
	double U = 2.0*pow(square, 14 / 2) - pow(square, 8 / 2);
	const double c = -24.0 * epsilon / sigmaSquare;
	mulc(r, c * U, F);
}


//==========================================================================
//					Operations with CUDAUnderspace
//==========================================================================


__device__ void d_recalculatePositions_Beeman(CUDAUnderspace *cus, double dt)
{
	for (size_t i = 0; i < cus->numberOfMolecules; ++i) {
		mov(cus->molecules[i].r, cus->molecules[i].oldr);
		CUDAVector tmp;
		//i.r += i.v * dt;
		mulc(cus->molecules[i].v, dt, tmp);
		add(cus->molecules[i].r, tmp, cus->molecules[i].r);
		//i.r += 4.0 / 6.0 * (i.F / i.m) * (dt*dt);
		mulc(cus->molecules[i].F, (4.0 / 6.0) * (dt*dt) / mass, tmp);
		add(cus->molecules[i].r, tmp, cus->molecules[i].r);
		//i.r -= 1.0 / 6.0 * (i.oldF / i.m) * (dt*dt);
		mulc(cus->molecules[i].oldF, - (1.0/6.0) * (dt*dt) / mass, tmp);
		add(cus->molecules[i].r, tmp, cus->molecules[i].r);
	}
}

__device__ void d_recalculateSpeeds_Beeman(CUDAUnderspace *cus, double dt, int width, int height)
{
	for (size_t i = 0; i < cus->numberOfMolecules; ++i) {
		//cus->molecules[i].newF.v[0] += 1e10;
		CUDAVector tmp;
		//i.v += 2.0 / 6.0 * (i.newF / i.m) * dt;
		mulc(cus->molecules[i].newF, (2.0 / 6.0) * dt / mass, tmp);
		add(cus->molecules[i].v, tmp, cus->molecules[i].v);
		//i.v += 5.0 / 6.0 * (i.F / i.m) * dt;
		mulc(cus->molecules[i].F, (5.0 / 6.0) * dt / mass, tmp);
		add(cus->molecules[i].v, tmp, cus->molecules[i].v);
		//i.v -= 1.0 / 6.0 * (i.oldF / i.m) * dt;
		mulc(cus->molecules[i].oldF, - (1.0 / 6.0) * dt / mass, tmp);
		add(cus->molecules[i].v, tmp, cus->molecules[i].v);

		if (cus->molecules[i].r.v[0] <= 0) {
			cus->molecules[i].v.v[0] = abs(cus->molecules[i].v.v[0]);
		}
		if (width * Angstrom <= cus->molecules[i].r.v[0]) {
			cus->molecules[i].v.v[0] = -abs(cus->molecules[i].v.v[0]);
		}

		if (cus->molecules[i].r.v[1] <= 0) {
			cus->molecules[i].v.v[1] = abs(cus->molecules[i].v.v[1]);
		}
		if (height * Angstrom <= cus->molecules[i].r.v[1]) {
			cus->molecules[i].v.v[1] = -abs(cus->molecules[i].v.v[1]);
		}
	}
}

__device__ void d_calculateNewForcesForUnderspace(CUDASpace *cs, int nx, int ny, int nz)
{
	CUDAUnderspace *centralSpace = &cs->underspaces[LINEAR(cs, nx, ny, nz)];
	int  closest = 1;
	for (int dx = -closest; dx <= closest; ++dx) {
		for (int dy = -closest; dy <= closest; ++dy) {
			for (int dz = -closest; dz <= closest; ++dz) {
				int x = nx + dx;
				int y = ny + dy;
				int z = nz + dz;
				if (x < 0 || y < 0 || z < 0) continue;
				if (x >= cs->Nx) continue;
				if (y >= cs->Ny) continue;
				if (z >= cs->Nz) continue;
				d_calculateNewForces(centralSpace, &cs->underspaces[LINEAR(cs, x, y, z)]);
			}
		}
	}
}

__device__ void d_calculateNewForces(CUDAUnderspace *cus1, CUDAUnderspace *cus2)
{
	for (size_t i = 0; i < cus1->numberOfMolecules; ++i) {
		for (size_t j = 0; j < cus2->numberOfMolecules; ++j) {
			//if (i == j && cus1 == cus2) continue;
			//Vector r = (*j).r - (*i).r;
			CUDAVector tmp;
			sub(cus2->molecules[j].r, cus1->molecules[i].r, tmp);
			double sq = square(tmp);
			if (sq == 0) continue;
			if (maxDistSquare < sq) continue;
			//(*i).newF += d_Force_LennardJones(r, sq);
			d_Force_LennardJones(tmp, sq, tmp);
			add(cus1->molecules[i].newF, tmp, cus1->molecules[i].newF);
		}
	}
}


//==========================================================================
//					Operations with CUDASpace
//==========================================================================

__global__ void cuda_recalculatePositions(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (nz >= cs->Nz) return;

	d_recalculatePositions_Beeman(&cs->underspaces[LINEAR(cs, nx, ny, nz)], cs->dt);
}

__global__ void cuda_recalculateSpeeds(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (nz >= cs->Nz) return;

	d_recalculateSpeeds_Beeman(&cs->underspaces[LINEAR(cs, nx, ny, nz)], cs->dt, cs->width, cs->height);
}

__global__ void cuda_recalculateForces(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;	
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (ny >= cs->Ny) return;

	d_calculateNewForcesForUnderspace(cs, nx, ny, nz);
}

__global__ void cuda_validate(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (nz >= cs->Nz) return;

	CUDAUnderspace& cus = cs->underspaces[LINEAR(cs, nx, ny, nz)];
	for (size_t i = 0; i < cus.numberOfMolecules; ++i) {
		//t.oldF = t.F;
		mov(cus.molecules[i].F, cus.molecules[i].oldF);
		//t.F = t.newF;
		mov(cus.molecules[i].newF, cus.molecules[i].F);
	}
}

__global__ void cuda_dropNewF(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (nz >= cs->Nz) return;
	CUDAUnderspace& cus = cs->underspaces[LINEAR(cs, nx, ny, nz)];
	for (size_t i = 0; i < cus.numberOfMolecules; ++i) {
		//t.newF = Vector();
		cus.molecules[i].newF.v[0] = 0;
		cus.molecules[i].newF.v[1] = 0;
		cus.molecules[i].newF.v[2] = 0;
	}
}

void cuda_oneStep(CUDASpace *d_cs,int Nx, int Ny, int Nz)
{
	int numberOfCores = 1024;
	//int coresPerDim = int(pow(numberOfCores, 1.0/3.0));
	int coresPerDim = int(sqrt(numberOfCores));
	dim3 grid, blocks;
	//grid = dim3(Nx / coresPerDim + 1, Ny / coresPerDim + 1, Nz / coresPerDim + 1);
	grid = dim3(Nx / coresPerDim + 1, Ny / coresPerDim + 1, Nz);
	blocks = dim3(coresPerDim, coresPerDim, 1);

	cuda_recalculatePositions	<<<grid, blocks>>> (d_cs);
	//auto cudaStatus = hipGetLastError();
	hipDeviceSynchronize();

	cuda_dropNewF				<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();

	cuda_recalculateForces		<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();

	cuda_recalculateSpeeds		<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();

	cuda_validate				<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();
}



extern CUDASpace* /*void*/ copyAndDeleteFromHost(CUDASpace *h_cs/*, CUDASpace **d_cs*/)
{
	CUDASpace *d_cs;
	hipMalloc(&d_cs, sizeof(CUDASpace));		//allocate memory for CUDASpace structure on device
	CUDAUnderspace *us_backup = h_cs->underspaces;		
	hipMalloc(&h_cs->underspaces, BYTES(h_cs));		//allocate memory for underspaces on device
	hipMemcpy(d_cs, h_cs, sizeof(CUDASpace), hipMemcpyHostToDevice);		//copy CUDASpace structure from h_cs (host) to d_cs (device)
	
	
	//allocate memory for molecules on device and copy arrays of molecules to device memory
	for (size_t i = 0; i < SIZE(h_cs); ++i) {
		CUDAMolecule *m_backup = us_backup[i].molecules;
		hipMalloc(&us_backup[i].molecules, sizeof(CUDAMolecule) * us_backup[i].numberOfMolecules);
		hipMemcpy(h_cs->underspaces + i, us_backup + i, sizeof(CUDAUnderspace), hipMemcpyHostToDevice);
		hipMemcpy(us_backup[i].molecules, m_backup, 
			sizeof(CUDAMolecule) * us_backup[i].numberOfMolecules, hipMemcpyHostToDevice);
		us_backup[i].molecules = m_backup;
	}
	h_cs->underspaces = us_backup;		//restore h_cs

	return d_cs;
}

CUDASpace* copyAndDeleteFromDevice(CUDASpace *d_cs/*, CUDASpace *h_cs*/)
{
	CUDASpace *h_cs = new CUDASpace;
	hipMemcpy(h_cs, d_cs, sizeof(CUDASpace), hipMemcpyDeviceToHost);
	CUDAUnderspace *d_cus = h_cs->underspaces;
	h_cs->underspaces = new CUDAUnderspace[BYTES(h_cs)];

	for (size_t i = 0; i < SIZE(h_cs); ++i) {
		hipMemcpy(h_cs->underspaces + i, d_cus + i, sizeof(CUDAUnderspace), hipMemcpyDeviceToHost);
		CUDAMolecule *d_cm = h_cs->underspaces[i].molecules;
		h_cs->underspaces[i].molecules = new CUDAMolecule[h_cs->underspaces[i].numberOfMolecules];
		hipMemcpy(h_cs->underspaces[i].molecules, d_cm, 
			sizeof(CUDAMolecule) * h_cs->underspaces[i].numberOfMolecules, hipMemcpyDeviceToHost);
	}

	return h_cs;
}

/*
void freeDeviceMem(CUDASpace *d_cs)
{
	for (size_t i = 0; i < d_cs->Nx; ++i) {
		for (size_t j = 0; j < d_cs->Ny; ++j) {
			for (size_t k = 0; k < d_cs->Nz; ++k) {
				hipFree(d_cs->underspaces[i][j][k].molecules);
			}
			hipFree(d_cs->underspaces[i][j]);
		}
		hipFree(d_cs->underspaces[i]);
	}
	hipFree(d_cs->underspaces);
	hipFree(d_cs);
}

void freeHostMem(CUDASpace *h_cs)
{
	for (size_t i = 0; i < h_cs->Nx; ++i) {
		for (size_t j = 0; j < h_cs->Ny; ++j) {
			for (size_t k = 0; k < h_cs->Nz; ++k) {
				delete h_cs->underspaces[i][j][k].molecules;
			}
			delete h_cs->underspaces[i][j];
		}
		delete h_cs->underspaces[i];
	}
	delete h_cs->underspaces;
	delete h_cs;
}
*/

