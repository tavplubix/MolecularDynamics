#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <iostream>

#include "c_cuda_structures.h"


//==========================================================================
//					Operations with CUDAVector
//==========================================================================

__device__ void mov(const CUDAVector &a, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i];
}

__device__ void add(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] + b.v[i];
}

__device__ void sub(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] - b.v[i];
}

__device__ void mulv(const CUDAVector &a, const CUDAVector& b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] * b.v[i];
}

__device__ void mulc(const CUDAVector &a, const double b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] * b;
}

__device__ void div(const CUDAVector &a, const double b, CUDAVector& result)
{
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result.v[i] = a.v[i] / b;
}

__device__ double square(const CUDAVector &a)
{
	double result = 0;
	for (size_t i = 0; i < VECTOR_DIMENSION; ++i)
		result += a.v[i] * a.v[i];
	return result;
}


//==========================================================================
//					Operations with CUDAMolecule
//==========================================================================

#define Boltzmann  1.3806488e-23
#define Angstrom  10e-10
#define AtomicMassUnit  1.660538921e-27
#define sigma 2.74*10e-10	//m
#define epsilon 36.2*1.3806488e-23	//J
#define mass 20.1797 * 1.660538921e-27

#define maxDistSquare 4.0 * sigma * 4.0 * sigma


__device__ inline void d_Force_LennardJones(const CUDAVector& r, double square, CUDAVector& F)
{
	const double sigmaSquare = sigma * sigma;
	square = sigmaSquare / square;
	double U = 2.0*pow(square, 14 / 2) - pow(square, 8 / 2);
	const double c = -24.0 * epsilon / sigmaSquare;
	mulc(r, c * U, F);
}


//==========================================================================
//					Operations with CUDAUnderspace
//==========================================================================

__device__ void d_recalculatePositions_Beeman(CUDAUnderspace *cus, double dt)
{
	for (size_t i = 0; i < cus->numberOfMolecules; ++i) {
		mov(cus->molecules[i].r, cus->molecules[i].oldr);
		CUDAVector tmp;
		//i.r += i.v * dt;
		mulc(cus->molecules[i].v, dt, tmp);
		add(cus->molecules[i].r, tmp, cus->molecules[i].r);
		//i.r += 4.0 / 6.0 * (i.F / i.m) * (dt*dt);
		mulc(cus->molecules[i].F, (4.0 / 6.0) * (dt*dt) / mass, tmp);
		add(cus->molecules[i].r, tmp, cus->molecules[i].r);
		//i.r -= 1.0 / 6.0 * (i.oldF / i.m) * (dt*dt);
		mulc(cus->molecules[i].oldF, - (1.0/6.0) * (dt*dt) / mass, tmp);
		add(cus->molecules[i].r, tmp, cus->molecules[i].r);
	}
}

__device__ void d_recalculateSpeeds_Beeman(CUDAUnderspace *cus, double dt, int width, int height)
{
	for (size_t i = 0; i < cus->numberOfMolecules; ++i) {
		CUDAVector tmp;
		//i.v += 2.0 / 6.0 * (i.newF / i.m) * dt;
		mulc(cus->molecules[i].newF, (2.0 / 6.0) * dt / mass, tmp);
		add(cus->molecules[i].v, tmp, cus->molecules[i].v);
		//i.v += 5.0 / 6.0 * (i.F / i.m) * dt;
		mulc(cus->molecules[i].F, (5.0 / 6.0) * dt / mass, tmp);
		add(cus->molecules[i].v, tmp, cus->molecules[i].v);
		//i.v -= 1.0 / 6.0 * (i.oldF / i.m) * dt;
		mulc(cus->molecules[i].oldF, - (1.0 / 6.0) * dt / mass, tmp);
		add(cus->molecules[i].v, tmp, cus->molecules[i].v);

		if (cus->molecules[i].r.v[0] <= 0) {
			cus->molecules[i].v.v[0] = abs(cus->molecules[i].v.v[0]);
		}
		if (width * Angstrom <= cus->molecules[i].r.v[0]) {
			cus->molecules[i].v.v[0] = -abs(cus->molecules[i].v.v[0]);
		}

		if (cus->molecules[i].r.v[1] <= 0) {
			cus->molecules[i].v.v[1] = abs(cus->molecules[i].v.v[1]);
		}
		if (height * Angstrom <= cus->molecules[i].r.v[1]) {
			cus->molecules[i].v.v[1] = -abs(cus->molecules[i].v.v[1]);
		}
	}
}

__device__ void d_calculateNewForcesForUnderspace(CUDASpace *cs, int nx, int ny, int nz)
{
	CUDAUnderspace *centralSpace = &cs->underspaces[nx][ny][nz];
	int  closest = 1;
	for (int dx = -closest; dx <= closest; ++dx) {
		for (int dy = -closest; dy <= closest; ++dy) {
			for (int dz = -closest; dz <= closest; ++dz) {
				int x = nx + dx;
				int y = ny + dy;
				int z = nz + dz;
				if (x < 0 || y < 0 || z < 0) continue;
				if (x >= cs->Nx) continue;
				if (y >= cs->Ny) continue;
				if (z >= cs->Nz) continue;
				d_calculateNewForces(centralSpace, &cs->underspaces[x][y][z]);
			}
		}
	}
}

__device__ void d_calculateNewForces(CUDAUnderspace *cus1, CUDAUnderspace *cus2)
{
	for (size_t i = 0; i < cus1->numberOfMolecules; ++i) {
		for (size_t j = 0; j < cus2->numberOfMolecules; ++j) {
			if (i == j) continue;
			//Vector r = (*j).r - (*i).r;
			CUDAVector tmp;
			sub(cus2->molecules[j].r, cus1->molecules[i].r, tmp);
			double sq = square(tmp);
			if (maxDistSquare < sq) continue;
			//(*i).newF += d_Force_LennardJones(r, sq);
			d_Force_LennardJones(tmp, sq, tmp);
			add(cus1->molecules[i].newF, tmp, cus1->molecules[i].newF);
		}
	}
}


//==========================================================================
//					Operations with CUDASpace
//==========================================================================
__global__ void cuda_recalculatePositions(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (ny >= cs->Ny) return;

	d_recalculatePositions_Beeman(&cs->underspaces[nx][ny][nz], cs->dt);
}

__global__ void cuda_recalculateSpeeds(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (ny >= cs->Ny) return;

	d_recalculateSpeeds_Beeman(&cs->underspaces[nx][ny][nz], cs->dt, cs->width, cs->height);
}

__global__ void cuda_recalculateForces(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	//if (nx >= cs->Nx) return;	
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	//if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	//if (ny >= cs->Ny) return;

	d_calculateNewForcesForUnderspace(cs, nx, ny, nz);
}

__global__ void cuda_validate(CUDASpace *cs)
{
	size_t nx = blockDim.x * blockIdx.x + threadIdx.x;
	if (nx >= cs->Nx) return;
	size_t ny = blockDim.y * blockIdx.y + threadIdx.y;
	if (ny >= cs->Ny) return;
	size_t nz = blockDim.z * blockIdx.z + threadIdx.z;
	if (ny >= cs->Ny) return;

	CUDAUnderspace& cus = cs->underspaces[nx][ny][nz];
	for (size_t i = 0; i < cus.numberOfMolecules; ++i) {
		//t.oldF = t.F;
		mov(cus.molecules[i].oldF, cus.molecules[i].F);
		//t.F = t.newF;
		mov(cus.molecules[i].F, cus.molecules[i].newF);
	}
}

void cuda_oneStep(CUDASpace *d_cs,int Nx, int Ny, int Nz)
{
	int numberOfCores = 512;
	int coresPerDim = numberOfCores / 3;
	dim3 grid, blocks;
	grid = dim3(Nx / coresPerDim + 1, Ny / coresPerDim + 1, Nz / coresPerDim + 1);
	blocks = dim3(coresPerDim, coresPerDim, coresPerDim);

	cuda_recalculatePositions	<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();
	cuda_recalculateForces		<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();
	cuda_recalculateSpeeds		<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();
	cuda_validate				<<<grid, blocks>>> (d_cs);
	hipDeviceSynchronize();
}



extern "C" CUDASpace* /*void*/ copyToDevice(CUDASpace *h_cs/*, CUDASpace **d_cs*/)
{
	CUDASpace *cs;
	hipMalloc(&cs, sizeof(CUDASpace));
	hipMalloc(&cs->underspaces, sizeof(CUDAUnderspace**) * h_cs->Nx);
	for (size_t i = 0; i < h_cs->Nx; ++i) {
		hipMalloc(&cs->underspaces[i], sizeof(CUDAUnderspace*) * h_cs->Ny);
		for (size_t j = 0; j < h_cs->Ny; ++j) {
			hipMalloc(&cs->underspaces[i][j], sizeof(CUDAUnderspace) * h_cs->Nz);
			for (size_t k = 0; k < h_cs->Nz; ++k) {
				hipMalloc(&cs->underspaces[i][j][k].molecules, sizeof(CUDAMolecule) * h_cs->underspaces[i][j][k].numberOfMolecules);
				hipMemcpy(cs->underspaces[i][j][k].molecules, h_cs->underspaces[i][j][k].molecules, h_cs->underspaces[i][j][k].numberOfMolecules, hipMemcpyHostToDevice);
				hipMemcpy(&cs->underspaces[i][j][k].numberOfMolecules, &h_cs->underspaces[i][j][k].numberOfMolecules, sizeof(h_cs->underspaces[i][j][k].numberOfMolecules), hipMemcpyHostToDevice);
			}
		}
	}
	hipMemcpy(&cs->dt, &h_cs->dt, sizeof(h_cs->dt), hipMemcpyHostToDevice);
	hipMemcpy(&cs->width, &h_cs->width, sizeof(h_cs->width), hipMemcpyHostToDevice);
	hipMemcpy(&cs->height, &h_cs->height, sizeof(h_cs->height), hipMemcpyHostToDevice);
	hipMemcpy(&cs->Nx, &h_cs->Nx, sizeof(h_cs->Nx), hipMemcpyHostToDevice);
	hipMemcpy(&cs->Ny, &h_cs->Ny, sizeof(h_cs->Ny), hipMemcpyHostToDevice);
	hipMemcpy(&cs->Nz, &h_cs->Nz, sizeof(h_cs->Nz), hipMemcpyHostToDevice);
	//*d_cs = cs;
	return cs;
}

CUDASpace* copyFromDevice(CUDASpace *d_cs/*, CUDASpace *h_cs*/)
{
	CUDASpace *cs = new CUDASpace;
	cs->underspaces = new CUDAUnderspace**[d_cs->Nx];
	for (size_t i = 0; i < d_cs->Nx; ++i) {
		cs->underspaces[i] = new CUDAUnderspace*[d_cs->Ny];
		for (size_t j = 0; j < d_cs->Ny; ++j) {
			cs->underspaces[i][j] = new CUDAUnderspace[d_cs->Nz];
			for (size_t k = 0; k < d_cs->Nz; ++k) {
				hipMalloc(&cs->underspaces[i][j][k].molecules, sizeof(CUDAMolecule) * d_cs->underspaces[i][j][k].numberOfMolecules);
				hipMemcpy(cs->underspaces[i][j][k].molecules, d_cs->underspaces[i][j][k].molecules, d_cs->underspaces[i][j][k].numberOfMolecules, hipMemcpyDeviceToHost);
				hipMemcpy(&cs->underspaces[i][j][k].numberOfMolecules, &d_cs->underspaces[i][j][k].numberOfMolecules, sizeof(d_cs->underspaces[i][j][k].numberOfMolecules), hipMemcpyDeviceToHost);
			}
		}
	}
	hipMemcpy(&cs->dt, &d_cs->dt, sizeof(d_cs->dt), hipMemcpyDeviceToHost);
	hipMemcpy(&cs->width, &d_cs->width, sizeof(d_cs->width), hipMemcpyDeviceToHost);
	hipMemcpy(&cs->height, &d_cs->height, sizeof(d_cs->height), hipMemcpyDeviceToHost);
	hipMemcpy(&cs->Nx, &d_cs->Nx, sizeof(d_cs->Nx), hipMemcpyDeviceToHost);
	hipMemcpy(&cs->Ny, &d_cs->Ny, sizeof(d_cs->Ny), hipMemcpyDeviceToHost);
	hipMemcpy(&cs->Nz, &d_cs->Nz, sizeof(d_cs->Nz), hipMemcpyDeviceToHost);

	return cs;
}

void freeDeviceMem(CUDASpace *d_cs)
{
	for (size_t i = 0; i < d_cs->Nx; ++i) {
		for (size_t j = 0; j < d_cs->Ny; ++j) {
			for (size_t k = 0; k < d_cs->Nz; ++k) {
				hipFree(d_cs->underspaces[i][j][k].molecules);
			}
			hipFree(d_cs->underspaces[i][j]);
		}
		hipFree(d_cs->underspaces[i]);
	}
	hipFree(d_cs->underspaces);
	hipFree(d_cs);
}

void freeHostMem(CUDASpace *h_cs)
{
	for (size_t i = 0; i < h_cs->Nx; ++i) {
		for (size_t j = 0; j < h_cs->Ny; ++j) {
			for (size_t k = 0; k < h_cs->Nz; ++k) {
				delete h_cs->underspaces[i][j][k].molecules;
			}
			delete h_cs->underspaces[i][j];
		}
		delete h_cs->underspaces[i];
	}
	delete h_cs->underspaces;
	delete h_cs;
}


